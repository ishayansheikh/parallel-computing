#include "hip/hip_runtime.h"
// C libraries
#include <stdio.h>
#include <stdlib.h>
// CUDA libraries
#include <hip/hip_runtime.h>
#include <>
// image processing library
#include "lodepng.h"

// Device Code
__global__ void rectification(unsigned char *input_image, unsigned char *output_image, int width, int height, int array_size) {
    // thread's x coordinate in the block, corresponds to width
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    // thread's y coordinate in the block, corresponds to height
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    if(x < width && y < height) {
        // calculate the index of the pixel in the input image array
        int pixel_index = 4 * width * y + 4 * x;
        // there are 4 values for a pixel: R, G, B, A. Loop over all of them to rectify them        
        for (int i = 0; i < 4; i++) {
            if (pixel_index + i < array_size) {
                int value = (int) input_image[pixel_index+i];
                if (value < 127) value = 127;
                output_image[pixel_index+i] = (unsigned char) value;
            }
        }
    }
}

// Host Code
int main(int argc, char *argv[]) {
    
    if (argc <= 1) {
        return printf("No arguments provided! Please add input file name, output file name and thread number to the program call!");
    } else if (argc > 1 && argc < 4) {
        return printf("Missing arguments! Please check that you have provided the input file name, output file name and the number of threads!");
    }

    // get inputs from the command line
    char *input_filename = argv[1];
    char *output_filename = argv[2];
    int threads_no = atoi(argv[3]);

    // initalize variables for error, input image, input image width and input image height
    unsigned error;
    unsigned char *input_image;
    unsigned width, height;

    // load input image from file to buffer array
    error = lodepng_decode32_file(&input_image, &width, &height, input_filename);
    
    // if there is an error while loading the file, return the error
    if(error) return printf("Error: %u: %s\n", error, lodepng_error_text(error));

    // initalize device variable to copy the input image over to the GPU
    unsigned char *d_input, *d_output;
    int size = width * height * 4 * sizeof(unsigned char);
    int array_size = width * height * 4;

    hipMalloc(&d_input, size);
    hipMalloc(&d_output, size);

    // create CUDA events to time the kernel runtime
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // copy image from host memory to device memory
    hipMemcpy(d_input, input_image, size, hipMemcpyHostToDevice);

    // initialize block size and block number to process the images
    dim3 blockSize(threads_no, threads_no);
    dim3 numBlocks(width/threads_no, height/threads_no);

    // record start time
    hipEventRecord(start);

    // run device kernel
    rectification<<<numBlocks, blockSize>>>(d_input, d_output, width, height, array_size);

    // record stop time
    hipEventRecord(stop);
    
    // synchronize device to get the output back from the device
    // hipDeviceSynchronize();

    // initialize output image array to copy output from device to host
    unsigned char *output_image = (unsigned char*)malloc(size);

    // copy output image from device to host
    hipMemcpy(output_image, d_output, size, hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // save output image
    lodepng_encode32_file(output_filename, output_image, width, height);

    // free up device memory;
    hipFree(d_input);
    hipFree(d_output);

    // free up host memory;
    free(output_image);

    //print elapsed time
    printf("Time Elapsed: %f ms\n", milliseconds);
    printf("Time Elapsed: %f ns\n", milliseconds * 1000);
}